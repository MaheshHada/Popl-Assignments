
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void blur_kernel(int *image_d,float *filter_d,int *blurimage_d,int N1,int N2) {
	int row = threadIdx.x + blockDim.x*blockIdx.x;
	int col = threadIdx.y + blockDim.y*blockIdx.y;
	if(row < N1 && col < N2) {
		int i,j;
		float sum = 0,wsum = 0;
		for(int i=0;i<3;i++) {
			for(j=0;j<3;j++) {
				wsum += *(filter_d + i*3 + j);
			}
		}
		int k = 0,l = 0;
		for(i=row-1;i<=row+1;i++,k++) {
			l = 0;
			for(j=col-1;j<=col+1;j++,l++) {
				if(i >= 0 && j >= 0 && i < N1 && j < N2) {
					sum += ((*(image_d + i*N2+j)) * (*(filter_d + k*3+l)));
				}
			}
		}
		blurimage_d[row*N2+col] = (int)(sum/wsum);
	}
}


int main() {
	int N1,N2;
	N1 = 100;
	N2 = 100;
	int *image_h = (int*)malloc(N1*N2*sizeof(int));
	int i,j;
	for(i=0;i<N1;i++) {
		for(j=0;j<N2;j++) {
			*(image_h + i*N2 + j) = rand()%256;
		}
	}
	float f[3][3] = {{1.0,2.0,1.0},{2.0,3.0,2.0},{1.0,2.0,1.0}};
	float *filter_h = (float*)malloc(3*3*sizeof(float));
	for(i=0;i<3;i++) {
		for(j=0;j<3;j++) {
			*(filter_h + i*3 + j) = f[i][j];
		}
	} 
	printf("Original Image:\n");
	for(i=0;i<N1;i++) {
		for(j=0;j<N2;j++) {
			printf("%d ",*(image_h + i*N2 + j));
		}
		printf("\n");	
	}
	printf("\n");
	int *image_d,*blurimage_d;
	float *filter_d;
	hipMalloc((void**)&image_d,N1*N2*sizeof(int));
	hipMalloc((void**)&filter_d,3*3*sizeof(float));
	hipMalloc((void**)&blurimage_d,N1*N2*sizeof(int));
	hipMemcpy(image_d,image_h,N1*N2*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(filter_d,filter_h,3*3*sizeof(float),hipMemcpyHostToDevice);
	dim3 grid(10,10);
	dim3 block(10,10);
	blur_kernel<<<grid,block>>>(image_d,filter_d,blurimage_d,N1,N2);
	hipMemcpy(image_h,blurimage_d,N1*N2*sizeof(int),hipMemcpyDeviceToHost);
	printf("Blurred Image:\n");
	for(i=0;i<N1;i++) {
		for(j=0;j<N2;j++) {
			printf("%d ",*(image_h + i*N2 + j));
		}
		printf("\n");	
	}
	printf("\n");
	free(image_h);
	free(filter_h);
	hipFree(image_d);
	hipFree(filter_d);
	hipFree(blurimage_d);
}
