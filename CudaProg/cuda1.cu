
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *a) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	a[idx] = 7;
}

int main() {
	int dimx = 16;
	int num_bytes = dimx*sizeof(int);
	int *d_a = 0,*h_a = 0;	//device and host paraemters
	h_a = (int*)malloc(num_bytes);
	hipMalloc((void**)&d_a,num_bytes);
	if( h_a == 0 || d_a == 0) {
		printf("Couldn't Allocate Memory\n");
		return 1;
	}
	hipMemset(d_a,0,num_bytes);
	dim3 grid,block;
	block.x = 4;
	grid.x = dimx/block.x;
	kernel<<<grid,block>>>(d_a);
	hipMemcpy(h_a,d_a,num_bytes,hipMemcpyDeviceToHost);
	for(int i=0;i<dimx;i++) {
		printf("%d",h_a[i]);
	}
	printf("\n");
	free(h_a);
	hipFree(d_a);
	return 0;
}