#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N1 3	
#define N2 3

__global__ void blur_kernel(int *image_d,float *filter_d,int *blurimage_d) {
	int row = threadIdx.x + blockDim.x*blockIdx.x;
	int col = threadIdx.y + blockDim.y*blockIdx.y;
	if(row < N1 && col < N2) {
		int i,j;
		float sum = 0,wsum = 0;
		for(int i=0;i<3;i++) {
			for(j=0;j<3;j++) {
				wsum += *(filter + i*3 + j);
			}
		}
		int k = 0,l = 0;
		for(i=row-1;i<=row+1;i++) {
			for(j=col-1;j<=col+1;j++) {
				if(i >= 0 && j >= 0 && i < N1 && j < N2) {
					sum += ((*(image_d + i*N2+j)) * (*(filter_d + k*3+l)));
				}
				l++;
			}
			k++;
		}
		blurimage_d[row*N2+col] = (int)(sum/wsum);
	}
}


int main() {
	//int N1,N2;
	//scanf("%d %d",&N1,&N2);
	int *image_h = (int*)malloc(N1*N2*sizeof(int));
	int i,j;
	for(i=0;i<N1;i++) {
		for(j=0;j<N2;j++) {
			*(image_h + i*N2 + j) = rand()%256;
		}
	}
	float f[3][3] = {{1.0,2.0,1.0},{2.0,3.0,2.0},{1.0,2.0,1.0}};
	float *filter_h = (float*)malloc(3*3*sizeof(float));
	for(i=0;i<3;i++) {
		for(j=0;j<3;j++) {
			*(filter_h + i*3 + j) = f[i][j];
		}
	} 
	for(i=0;i<N1;i++) {
		for(j=0;j<N2;j++) {
			printf("%d ",*(image_h + i*N2 + j));
		}
		printf("\n");	
	}
	printf("\n");
	for(i=0;i<3;i++) {
		for(j=0;j<3;j++) {
			printf("%d ",*(filter_h + i*3 + j));
		}
		printf("\n");	
	}
	printf("\n");
	int *image_d,*blurimage_d;
	float *filter_d;
	hipMalloc((void**)&image_d,N1*N2*sizeof(int));
	hipMalloc((void**)&filter_d,3*3*sizeof(float));
	hipMalloc((void**)&blurimage_d,N1*N2*sizeof(int));
	hipMemcpy(image_d,image_h,N1*N2*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(filter_d,filter_h,3*3*sizeof(float),hipMemcpyHostToDevice);
	dim3 grid(N2,N1);
	dim3 block(1,1);
	blur_kernel<<<grid,block>>>(image_d,filter_d,blurimage_d);
	hipMemcpy(image_h,blurimage_d,N1*N2*sizeof(int),hipMemcpyDeviceToHost);
	for(i=0;i<N1;i++) {
		for(j=0;j<N2;j++) {
			printf("%d ",*(image_h + i*N2 + j));
		}
		printf("\n");	
	}
	printf("\n");
	free(image_h);
	free(filter_h);
	hipFree(image_d);
	hipFree(filter_d);
	hipFree(blurimage_d);
}
