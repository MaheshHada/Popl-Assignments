
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(float *A,float *B,float *C) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	C[i] = A[i] + B[i];
}

int main() {
	int N;
	scanf("%d",&N);
	float *h_a = (float*)malloc(N*sizeof(float));
	float *h_b = (float*)malloc(N*sizeof(float));
	float *h_c = (float*)malloc(N*sizeof(float));
	float *d_a,*d_b,*d_c;
	hipMalloc((void**)&d_a,N*sizeof(float));
	hipMalloc((void**)&d_b,N*sizeof(float));
	hipMalloc((void**)&d_c,N*sizeof(float));
	int i;
	for(i=0;i<N;i++) {
		*(h_a + i) = (float)rand()/(float)(RAND_MAX/100);
	}
	for(i=0;i<N;i++) {
		*(h_b + i) = (float)rand()/(float)(RAND_MAX/100);
	}
	for(i=0;i<N;i++) {
		*(h_c + i) = (float)rand()/(float)(RAND_MAX/100);
	}
	hipMemcpy(d_a,h_a,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemset(d_c,0,N*sizeof(float));
	vecAdd<<<N/256,256>>>(d_a,d_b,d_c);
	hipMemcpy(h_c,d_c,N*sizeof(float),hipMemcpyDeviceToHost);
	for(i=0;i<N;i++) {
		printf("%d ",h_a[i]);
	}
	printf("\n");
	for(i=0;i<N;i++) {
		printf("%d ",h_b[i]);
	}
	printf("\n");
	for(i=0;i<N;i++) {
		printf("%d ",h_c[i]);
	}
	printf("\n");
	free(h_a);
	free(h_b);
	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}